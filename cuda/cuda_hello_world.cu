
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU! %d\n", threadIdx.x*gridDim.x);
}

int main() {
    printf("Hello World from CPU!\n");
    hipSetDevice(0);
    cuda_hello<<<1,10>>>();
    uint8_t *cuda_mem_;
    if (hipMalloc(&cuda_mem_, 1024) != hipSuccess)
    {
        throw std::runtime_error("Failed to allocate device memory");
    }

    hipDeviceSynchronize();
    return 0;
}